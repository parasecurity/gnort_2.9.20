# include <stdio.h>
#include <hip/hip_runtime.h>



/*
*  Text Data Buffer
*/ 
static unsigned char text[512];
static const char *text_ptr = (const char*)text;

ACCUDA_STRUCT * acNew () {
  // print_packet_buffer_info();

  // if (isStreamInitialized == 0) {
	//   hipStreamCreate(&stream);
  //   #ifdef DEBUG
  //   CUT_CHECK_ERROR("hipStreamCreate");
  //   #endif
  //   isStreamInitialized = 1;
}

void printCommandInfo(int n, const char ** args){
    for (int i = 0; i < n; i++){
      printf("ARGUEMENT %d: %s\n", i, args[i]);
    }
}
#define ACCUDA_MAIN // uncomment this to run individually: nvcc -o ac_cuda ../src/sfutil/ac_cuda.cu
#ifdef ACCUDA_MAIN

int main (int argc, const char **argv){
  int i, nocase = 0;

  if (argc < 4)
  {
    fprintf (stderr,
      "Usage: ./ac_cuda acx text pattern-1 pattern-2 ... pattern-n  -nocase\n");
      // ---->argv[0], v[1] v[2]    v[3]      v[4]   ...   v[n+2]       v[n+3]
    exit (0);
  }

  // printCommandInfo(argc, argv);
  strcpy ((char *)text, argv[2]);
  for (int i = 2; i < argc; i++)
    if (strcmp (argv[i], "-nocase") == 0)
      nocase = 1;
  printf("Text string: %s\n", text);


  for (i = 3; i < argc; i++)
  {
    if (argv[i][0] == '-')
      continue;
    memcpy ((char *)text, argv[i], strlen(argv[i]) + 1);
    printf("Pattern[%d]: %s\n",i-2 , text);
    //acAddPattern (ac, (unsigned char*)text, strlen (text_ptr), nocase, 0, 0, (unsigned char*)argv[i], i - 2);
  }

}

#endif

