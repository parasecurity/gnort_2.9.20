#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>

#include <sys/time.h>
#include <time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <assert.h>
// #define ACCUDA_MAIN // uncomment this to run individually: nvcc -o ac_cuda ../src/sfutil/ac_cuda.cu
#ifndef ACCUDA_MAIN
#include "/home/zxcv/snort_2.9.20/snort-2.9.20/src/snort.h" // for PacketCount struct
#endif

#include <sys/types.h>
#include <sys/ipc.h>
#include <sys/sem.h>

#include "ac_cuda.cu.h"

#define MEM_OPTS
#ifdef MEM_OPTS
 #ifdef TEX_STATE_TABLE
 static texture <acstate_t, 1, hipReadModeElementType> texStateTable;
 #endif
#else
static texture <acstate_t, 2, hipReadModeElementType> texStateTable;
#endif

/*
*  Text Data Buffer
*/ 
static unsigned char text[512];
static const char *text_ptr = (const char*)text;

extern unsigned int UsePinnedMem;
#ifdef MEM_OPTS
  typedef struct packet_buffer_ {
    unsigned char bufNo; // 0 or 1
    unsigned char buf[2][BUFFER_SIZE][PKT_SNAPLEN];
    unsigned int bufPos;
  } packet_buffer_t;

  static Packet pktHdr[2][BUFFER_SIZE];

  static PORT_GROUP *pktPg[2][BUFFER_SIZE];

  static packet_buffer_t *packetBuffer = 0x00;

  static acstate_t *stateTable = NULL;
  static unsigned int stateTablePos = 0;


  static hipStream_t stream;
  static unsigned int isStreamInitialized = 0;
  #ifdef TEX_STATE_TABLE
    static unsigned int stateTableSize = 131072; // 128MB; max number of states that can be bound to texture
  #else
    static unsigned int stateTableSize = 1048576; // 1GB;
  #endif

#endif


static void print_packet_buffer_info() {
  static int first_time=1;

  if (first_time) {
    first_time = 0;

    fprintf(stdout, "Packet buffer [%s]: %d packets, %d bytes each (Total: %d bytes)\n",
                        (UsePinnedMem) ? "PINNED" : "PAGEABLE",
                        BUFFER_SIZE, PKT_SNAPLEN, BUFFER_SIZE * PKT_SNAPLEN);
    fflush(stdout);
  }
}

ACCUDA_STRUCT * acNew () {
  // print_packet_buffer_info();

  // if (isStreamInitialized == 0) {
	//   hipStreamCreate(&stream);
  //   #ifdef DEBUG
  //     CUT_CHECK_ERROR("hipStreamCreate");
  //   #endif
  //   isStreamInitialized = 1;
  // }
}

void printCommandInfo(int n, const char ** args){
    for (int i = 0; i < n; i++){
      printf("ARGUEMENT %d: %s\n", i, args[i]);
    }
}
//#define ACCUDA_MAIN 
#ifdef ACCUDA_MAIN

int main (int argc, const char **argv){
  int i, nocase = 0;
  ACCUDA_STRUCT * ac;

  if (argc < 4)
  {
    fprintf (stderr,
      "Usage: ./ac_cuda acx text pattern-1 pattern-2 ... pattern-n  -nocase\n");
      // ---->argv[0], v[1] v[2]    v[3]      v[4]   ...   v[n+2]       v[n+3]
    exit (0);
  }

  // printCommandInfo(argc, argv);
  strcpy ((char *)text, argv[2]);
  for (int i = 2; i < argc; i++)
    if (strcmp (argv[i], "-nocase") == 0)
      nocase = 1;
  printf("Text string: %s\n", text);

  ac = acNew();


  for (i = 3; i < argc; i++)
  {
    if (argv[i][0] == '-')
      continue;
    memcpy ((char *)text, argv[i], strlen(argv[i]) + 1);
    printf("Pattern[%d]: %s\n",i-2 , text);
    //acAddPattern (ac, (unsigned char*)text, strlen (text_ptr), nocase, 0, 0, (unsigned char*)argv[i], i - 2);
  }

}

#endif

